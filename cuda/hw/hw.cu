
#include <hip/hip_runtime.h>
#include <cstdio>


__global__ void helloFromGPU() {
    printf("Hello from GPU thread %i!\n", threadIdx.x);
}

int main() {

    printf("Hello from CPU!\n");

    helloFromGPU<<<1, 10>>>();

    hipDeviceSynchronize();

    return 0;
}
